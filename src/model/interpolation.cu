#include "hip/hip_runtime.h"
/*
 * interpolation.cu
 *
 *  Created on: Sep 4, 2016
 *      Author: uwe
 */

#include "interpolation.h"

#include "nativeTypesWrapper.h"
#include "DeviceIntrplGrid.h"
#include "DeviceProtein.h"
#include "macros.h"
#include "math_functions.h"

namespace as {

template <typename T>
__host__ __device__
__forceinline__ T lerp(T v0, T v1, T t) {
    return fma(t, v1, fma(-t, v0, v0));
}


__host__ __device__
__forceinline__ float4 lerp4f(float4 v0, float4 v1, float t) {
    return make_float4( lerp<float>(v0.x, v1.x, t), lerp<float>(v0.y, v1.y, t), lerp<float>(v0.z, v1.z, t), lerp<float>(v0.w, v1.w, t) );
}


template<typename REAL>
__host__ __device__
__forceinline__ float4 interpolate2( const d_IntrlpGrid<REAL>& grid,unsigned const type, REAL x, REAL y, REAL z, unsigned const i){
//TEMP
//	unsigned idxX1 = (unsigned) floor(
//				(x - grid.minDim.x) * grid.dVox_inv);
//		unsigned idxY1 = (unsigned) floor(
//				(y - grid.minDim.y) * grid.dVox_inv);
//		unsigned idxZ1 = (unsigned) floor(
//				(z - grid.minDim.z) * grid.dVox_inv);
//
//		// compute absolute position of the vertices
//		VoxelOctet<float> voxelOct;
//		voxelOct.min.x = idxX1 * grid.dVox + grid.minDim.x;
//		voxelOct.min.y = idxY1 * grid.dVox + grid.minDim.y;
//		voxelOct.min.z = idxZ1 * grid.dVox + grid.minDim.z;
//		voxelOct.max.x = voxelOct.min.x + grid.dVox;
//		voxelOct.max.y = voxelOct.min.y + grid.dVox;
//		voxelOct.max.z = voxelOct.min.z + grid.dVox;
//		voxelOct.data[0][0][0] = tex3D<float4>(grid.texArrayPt[type], idxX1, idxY1, idxZ1);
//		/** use of non-normalized coordinates */
//
//		float idxNx1 =  0.5 +(float) idxX1;  //
//		float idxNy1 =  0.5 +(float) idxY1;  //
//		float idxNz1 =  0.5 +(float) idxZ1;  //
//
//		float3 pos = make_float3(x, y, z);
//			float3 pos_m_posMin ;
//			pos_m_posMin.x = pos.x - voxelOct.min.x;
//			pos_m_posMin.y = pos.y - voxelOct.min.y;
//			pos_m_posMin.z = pos.z - voxelOct.min.z;
//			float3 posMax_m_pos ;
//			posMax_m_pos.x = voxelOct.max.x - pos.x;
//			posMax_m_pos.y = voxelOct.max.y - pos.y;
//			posMax_m_pos.z = voxelOct.max.z - pos.z;
//
//			float tmpMax_xy = (posMax_m_pos.x) * (posMax_m_pos.y);
//			float tmpMin_xy = (pos_m_posMin.x) * (pos_m_posMin.y);
//
//			float4 V = voxelOct.data[0][0][0] * (tmpMax_xy * (posMax_m_pos.z))
//					+ voxelOct.data[1][0][0]
//							* ((pos_m_posMin.x) * (posMax_m_pos.y) * (posMax_m_pos.z))
//					+ voxelOct.data[0][1][0]
//							* ((posMax_m_pos.x) * (pos_m_posMin.y) * (posMax_m_pos.z))
//					+ voxelOct.data[0][0][1] * (tmpMax_xy * (pos_m_posMin.z))
//					+ voxelOct.data[1][0][1]
//							* ((pos_m_posMin.x) * (posMax_m_pos.y) * (pos_m_posMin.z))
//					+ voxelOct.data[0][1][1]
//							* ((posMax_m_pos.x) * (pos_m_posMin.y) * (pos_m_posMin.z))
//					+ voxelOct.data[1][1][0] * (tmpMin_xy * (posMax_m_pos.z))
//					+ voxelOct.data[1][1][1] * (tmpMin_xy * (pos_m_posMin.z));



		//TEMP
	x = (x - grid.minDim.x) * grid.dVox_inv;
	y = (y - grid.minDim.y) * grid.dVox_inv;
	z = (z - grid.minDim.z) * grid.dVox_inv;

	unsigned const idxX = (unsigned) floor(x);
	unsigned const idxY = (unsigned) floor(y);
	unsigned const idxZ = (unsigned) floor(z);

	REAL const a = x - (REAL)idxX;
	REAL const b = y - (REAL)idxY;
	REAL const c = z - (REAL)idxZ;
	float4 data[2][2][2];
	data[0][0][0] = tex3D<float4>(grid.texArrayPt[type], idxX, idxY, idxZ);
	data[0][0][1] = tex3D<float4>(grid.texArrayPt[type], idxX, idxY, idxZ + 1);
	data[0][1][1] = tex3D<float4>(grid.texArrayPt[type], idxX, idxY + 1, idxZ + 1);
	data[0][1][0] = tex3D<float4>(grid.texArrayPt[type], idxX, idxY + 1, idxZ);
	data[1][1][0] = tex3D<float4>(grid.texArrayPt[type], idxX + 1, idxY + 1, idxZ);
	data[1][1][1] = tex3D<float4>(grid.texArrayPt[type], idxX + 1, idxY + 1, idxZ + 1);
	data[1][0][1] = tex3D<float4>(grid.texArrayPt[type], idxX + 1, idxY, idxZ + 1);
	data[1][0][0] = tex3D<float4>(grid.texArrayPt[type], idxX + 1, idxY, idxZ);

	float4 result =	lerp4f(
					lerp4f(
						lerp4f(data[0][0][0],data[0][0][1],c),
						lerp4f(data[0][1][0],data[0][1][1],c),
						b),
					lerp4f(
						lerp4f(data[1][0][0],data[1][0][1],c),
						lerp4f(data[1][1][0],data[1][1][1],c),
						b),
					a);


	//printf("%d %d %d %d     %f %f %f\n",i,idxX,idxY,idxZ,idxNx1,idxNy1,idxNz1);
	//printf("%d %f \n",i,data[0][0][0].x - voxelOct.data[0][0][0].x);
	//printf("%d %f %f\n",i,result.x, V.x);


return result;


}




 template <typename REAL>
 __forceinline__ __device__ void getVoxelDevice(const d_IntrlpGrid<REAL>& grid, const unsigned &type,
		const float &x, const float &y,	const float &z,  VoxelOctet<float>& voxelOct, unsigned const i)
{
	unsigned idxX = (unsigned) floor(
			(x - grid.minDim.x) * grid.dVox_inv);
	unsigned idxY = (unsigned) floor(
			(y - grid.minDim.y) * grid.dVox_inv);
	unsigned idxZ = (unsigned) floor(
			(z - grid.minDim.z) * grid.dVox_inv);

	// compute absolute position of the vertices
	voxelOct.min.x = idxX * grid.dVox + grid.minDim.x;
	voxelOct.min.y = idxY * grid.dVox + grid.minDim.y;
	voxelOct.min.z = idxZ * grid.dVox + grid.minDim.z;
	voxelOct.max.x = voxelOct.min.x + grid.dVox;
	voxelOct.max.y = voxelOct.min.y + grid.dVox;
	voxelOct.max.z = voxelOct.min.z + grid.dVox;

	/** use of non-normalized coordinates */

	float idxNx =  0.5 +(float) idxX;  //
	float idxNy =  0.5 +(float) idxY;  //
	float idxNz =  0.5 +(float) idxZ;  //
	//printf("%d %f %f %f\n",i,idxNx,idxNy,idxNz);
	//printf("%d \t %0.1f %0.1f %0.1f \t %f %f %f %f\n",i,idxNx,idxNy,idxNz,voxelOct.data[0][0][0].x,voxelOct.data[0][0][0].y,voxelOct.data[0][0][0].z,voxelOct.data[0][0][0].w );
	voxelOct.data[0][0][0] = tex3D<float4>(grid.texArrayPt[type], idxNx, idxNy, idxNz);
	voxelOct.data[1][0][0] = tex3D<float4>(grid.texArrayPt[type], idxNx + 1, idxNy, idxNz);
	voxelOct.data[0][1][0] = tex3D<float4>(grid.texArrayPt[type], idxNx, idxNy + 1, idxNz);
	voxelOct.data[1][1][0] = tex3D<float4>(grid.texArrayPt[type], idxNx + 1, idxNy + 1, idxNz);

	voxelOct.data[0][0][1] = tex3D<float4>(grid.texArrayPt[type], idxNx, idxNy, idxNz + 1);
	voxelOct.data[1][0][1] = tex3D<float4>(grid.texArrayPt[type], idxNx + 1, idxNy, idxNz + 1);
	voxelOct.data[0][1][1] = tex3D<float4>(grid.texArrayPt[type], idxNx, idxNy + 1, idxNz + 1);
	voxelOct.data[1][1][1] = tex3D<float4>(grid.texArrayPt[type], idxNx + 1, idxNy + 1, idxNz + 1);

}


/*
 ** @brief: function body for a trilinear interpolation.
 */

 __forceinline__ __host__ __device__ void trilinearInterpolation(const float &x,
		const float &y, const float &z, const VoxelOctet<float> &voxelOct,
		const float &voxelVol_inv, float4 &V)
{
	/* for operator overloading of *,+,-,/ for cuda types (REAL4)
	 * they are defined in asUtils/cudaMath*/

	 //printf("%.20f %.20f %.20f\n",x,y,z);
	float3 pos = make_float3(x, y, z);
	float3 pos_m_posMin ;
	pos_m_posMin.x = pos.x - voxelOct.min.x;
	pos_m_posMin.y = pos.y - voxelOct.min.y;
	pos_m_posMin.z = pos.z - voxelOct.min.z;
	float3 posMax_m_pos ;
	posMax_m_pos.x = voxelOct.max.x - pos.x;
	posMax_m_pos.y = voxelOct.max.y - pos.y;
	posMax_m_pos.z = voxelOct.max.z - pos.z;

	float tmpMax_xy = (posMax_m_pos.x) * (posMax_m_pos.y);
	float tmpMin_xy = (pos_m_posMin.x) * (pos_m_posMin.y);

	V = voxelOct.data[0][0][0] * (tmpMax_xy * (posMax_m_pos.z))
			+ voxelOct.data[1][0][0]
					* ((pos_m_posMin.x) * (posMax_m_pos.y) * (posMax_m_pos.z))
			+ voxelOct.data[0][1][0]
					* ((posMax_m_pos.x) * (pos_m_posMin.y) * (posMax_m_pos.z))
			+ voxelOct.data[0][0][1] * (tmpMax_xy * (pos_m_posMin.z))
			+ voxelOct.data[1][0][1]
					* ((pos_m_posMin.x) * (posMax_m_pos.y) * (pos_m_posMin.z))
			+ voxelOct.data[0][1][1]
					* ((posMax_m_pos.x) * (pos_m_posMin.y) * (pos_m_posMin.z))
			+ voxelOct.data[1][1][0] * (tmpMin_xy * (posMax_m_pos.z))
			+ voxelOct.data[1][1][1] * (tmpMin_xy * (pos_m_posMin.z));

	V = V * voxelVol_inv;
	return;
}
template<typename REAL>
__forceinline__ __device__ float4 Intrpl3D(const d_IntrlpGrid<REAL>& grid, const unsigned& type, const float &x, const float &y,
		const float &z, unsigned const i)
{

	VoxelOctet<float> voxelOct;
	getVoxelDevice<REAL>(grid, type, x, y, z, voxelOct,i);
	float4 V;
	trilinearInterpolation(x, y, z, voxelOct, grid.voxelVol_inv, V);
	return V;
}

template<typename REAL>
__global__ void d_innerPotForce (
		const d_IntrlpGrid<REAL> grid,
		const d_Protein<REAL> prot,
		const unsigned numDOFs,
		const REAL* data_in_x,
		const REAL* data_in_y,
		const REAL* data_in_z,
		REAL* data_out_x,
		REAL* data_out_y,
		REAL* data_out_z,
		REAL* data_out_E)
{
	using real4_t = typename TypeWrapper<REAL>::real4_t;
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned numAtoms = prot.numAtoms;
	if (idx < numAtoms*numDOFs) {
		unsigned type = prot.mappedType[idx % numAtoms];
		float4 pot = {0,0,0,0};
		if (type != 0) {

			REAL x = data_in_x[idx];
			REAL y = data_in_y[idx];
			REAL z = data_in_z[idx];
			bool test =false;

			if ((x >= grid.minDim.x && x <= grid.maxDim.x)
					&& (y >= grid.minDim.y && y <= grid.maxDim.y)
					&& (z >= grid.minDim.z && z <= grid.maxDim.z))
			{
				if(test){

					x = (x - grid.minDim.x) * grid.dVox_inv + 0.5;
					y = (y - grid.minDim.y) * grid.dVox_inv + 0.5;
					z = (z - grid.minDim.z) * grid.dVox_inv + 0.5;

					pot = tex3D<float4>(grid.texArrayLin[type], x, y, z); /** Interpolated value */

					REAL charge = prot.charge[idx % numAtoms];
					if (fabs(charge) > 0.001f) {
						float4 V_el = tex3D<float4>(grid.texArrayLin[0], x, y, z); /** Interpolated value */
						pot = pot + V_el * charge;
					}
				}else{

					//pot = interpolate2<REAL>(  grid,type,  x, y,  z,idx);
					pot = Intrpl3D<REAL>(grid, type, (float)x,  (float)y,  (float)z,idx);

					//printf("%f %f\n", pot.y,test.y);
					//pot = tex3Dfetch<float4>(grid.texArrayLin[type], {idxX, idxY, idxZ,1}); /** Interpolated value */

					REAL charge = prot.charge[idx % numAtoms];
					if (fabs(charge) > 0.001f) {

						float4 V_el = Intrpl3D<REAL>(grid, 0, x, y, z,idx);
						//float4 V_el= interpolate2<REAL>(  grid,0,  x, y,  z,idx);
						pot = pot + V_el * charge;
					}
				}
			}
		}

		data_out_x[idx] = static_cast<REAL>(pot.x);
		data_out_y[idx] = static_cast<REAL>(pot.y);
		data_out_z[idx] = static_cast<REAL>(pot.z);
		data_out_E[idx] = static_cast<REAL>(pot.w);
	}
}
template<typename REAL>
__device__ __forceinline__ void PotForce_device(
		const d_IntrlpGrid<REAL> inner,
		const d_IntrlpGrid<REAL> outer,
		const d_Protein<REAL> prot,
		const unsigned numDOFs,
		const unsigned idx,
		const REAL x,
		const REAL y,
		const REAL z,
		float4 & data_out
		)
{

	using real4_t = typename TypeWrapper<REAL>::real4_t;


	const unsigned numAtoms = prot.numAtoms;
	unsigned type = prot.mappedType[idx % numAtoms];
	REAL charge = prot.charge[idx % numAtoms];
	if (type != 0) {
		if ((x >= inner.minDim.x && x <= inner.maxDim.x)
		 && (y >= inner.minDim.y && y <= inner.maxDim.y)
		 && (z >= inner.minDim.z && z <= inner.maxDim.z)){
			 gridForce( inner, x, y, z, type,charge, data_out);
		}

		else if ( ((x >= outer.minDim.x && x <= outer.maxDim.x)
				&& (y >= outer.minDim.y && y <= outer.maxDim.y)
				&& (z >= outer.minDim.z && z <= outer.maxDim.z)))
		{
			gridForce( outer, x, y, z, type,charge, data_out);
		}
	}
}


template<typename REAL>
__device__ __forceinline__ void gridForce(
		const d_IntrlpGrid<REAL> grid,
		 REAL x,
		 REAL y,
		 REAL z,
		const unsigned type,
		REAL charge,
		float4& data_out
		)
{

	using real4_t = typename TypeWrapper<REAL>::real4_t;
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;
	bool test =false;
	if(test){
		x = (x - grid.minDim.x) * grid.dVox_inv + 0.5f;
		y = (y - grid.minDim.y) * grid.dVox_inv + 0.5f;
		z = (z - grid.minDim.z) * grid.dVox_inv + 0.5f;
		data_out = tex3D<float4>(grid.texArrayLin[type], x, y, z); /** Interpolated value */


		if (fabs(charge) > 0.001f) {
			float4 V_el = tex3D<float4>(grid.texArrayLin[0], x, y, z); /** Interpolated value */
			data_out = data_out + V_el * charge;
		}
	}else{
		data_out = Intrpl3D<REAL>( grid, type, (float)x,  (float)y,  (float)z,idx);
		//pot = interpolate2<REAL>(  outer, type,  x, y,  z,idx);
		//REAL charge = prot.charge[idx % numAtoms];
		if (fabs(charge) > 0.001f) {

			float4 V_el = Intrpl3D<REAL>( grid, 0, x, y, z,idx);
			//float4 V_el = interpolate2<REAL>(  outer, 0,  x, y,  z,idx);
			data_out = data_out + V_el * charge;
		}
	}
}


template<typename REAL>
__global__ void d_outerPotForce(
		const d_IntrlpGrid<REAL> inner,
		const d_IntrlpGrid<REAL> outer,
		const d_Protein<REAL> prot,
		const unsigned numDOFs,
		const REAL* data_in_x,
		const REAL* data_in_y,
		const REAL* data_in_z,
		REAL* data_out_x,
		REAL* data_out_y,
		REAL* data_out_z,
		REAL* data_out_E)
{

	using real4_t = typename TypeWrapper<REAL>::real4_t;
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;

	//DEBUG
//	if (idx == 0) {
//		printf("%f %f %f %f %f %f\n" ,
//				grid.minDim.x, grid.minDim.y, grid.minDim.z,
//				grid.maxDim.x, grid.maxDim.y, grid.maxDim.z);
//	}


	const unsigned numAtoms = prot.numAtoms;
	if (idx < numAtoms*numDOFs) {
		unsigned type = prot.mappedType[idx % numAtoms];
		if (type != 0) {

			REAL x = data_in_x[idx];
			REAL y = data_in_y[idx];
			REAL z = data_in_z[idx];

//			if (idx < 50) {
//				printf("%u %f %f %f %u\n" ,
//						idx, x, y, z, type);
//			}

			if (      ((x < inner.minDim.x || x > inner.maxDim.x)
					|| (y < inner.minDim.y || y > inner.maxDim.y)
					|| (z < inner.minDim.z || z > inner.maxDim.z))
					&&
					  ((x >= outer.minDim.x && x <= outer.maxDim.x)
					&& (y >= outer.minDim.y && y <= outer.maxDim.y)
					&& (z >= outer.minDim.z && z <= outer.maxDim.z)))
			{
				bool test =false;
				float4 pot{0,0,0,0};
				if(test){
				x = (x - outer.minDim.x) * outer.dVox_inv + 0.5f;
				y = (y - outer.minDim.y) * outer.dVox_inv + 0.5f;
				z = (z - outer.minDim.z) * outer.dVox_inv + 0.5f;

				pot = tex3D<float4>(outer.texArrayLin[type], x, y, z); /** Interpolated value */

				REAL charge = prot.charge[idx % numAtoms];
				if (fabs(charge) > 0.001f) {
					float4 V_el = tex3D<float4>(outer.texArrayLin[0], x, y, z); /** Interpolated value */
					pot = pot + V_el * charge;
				}

//				if (idx < 20) {
//					printf("%u %f %f %f %f %f %f\n" ,
//							idx, pot.x, pot.y, pot.z, pot.w);
//				}
				}else{


					pot = Intrpl3D<REAL>(outer, type, (float)x,  (float)y,  (float)z,idx);
					//pot = interpolate2<REAL>(  outer, type,  x, y,  z,idx);


					REAL charge = prot.charge[idx % numAtoms];
					if (fabs(charge) > 0.001f) {

						float4 V_el = Intrpl3D<REAL>(outer, 0, x, y, z,idx);
						//float4 V_el = interpolate2<REAL>(  outer, 0,  x, y,  z,idx);
						pot = pot + V_el * charge;
					}
				}

				data_out_x[idx] = static_cast<REAL>(pot.x);
				data_out_y[idx] = static_cast<REAL>(pot.y);
				data_out_z[idx] = static_cast<REAL>(pot.z);
				data_out_E[idx] = static_cast<REAL>(pot.w);
			}
		}
	}
}

template<typename REAL>
void d_potForce (
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		const d_IntrlpGrid<REAL>& inner,
		const d_IntrlpGrid<REAL>& outer,
		const d_Protein<REAL>& prot,
		const unsigned& numDOFs,
		const REAL* data_in_x,
		const REAL* data_in_y,
		const REAL* data_in_z,
		REAL* data_out_x,
		REAL* data_out_y,
		REAL* data_out_z,
		REAL* data_out_E)
{
	cudaVerifyKernel((
			d_innerPotForce<<<gridSize, blockSize, 0, stream>>> (
				inner,
				prot,
				numDOFs,
				data_in_x,
				data_in_y,
				data_in_z,
				data_out_x,
				data_out_y,
				data_out_z,
				data_out_E
			)
		));

	cudaVerifyKernel((
			d_outerPotForce<<<gridSize, blockSize, 0, stream>>> (
				inner,
				outer,
				prot,
				numDOFs,
				data_in_x,
				data_in_y,
				data_in_z,
				data_out_x,
				data_out_y,
				data_out_z,
				data_out_E
			)
		));
}

template
void d_potForce<float> (
	unsigned blockSize,	unsigned gridSize, const hipStream_t &stream,
	const d_IntrlpGrid<float>& inner, const d_IntrlpGrid<float>& outer, const d_Protein<float>& prot,
	const unsigned& numDOFs,
	const float* data_in_x, const float* data_in_y, const float* data_in_z,
	float* data_out_x, float* data_out_y, float* data_out_z, float* data_out_E);

template
void d_potForce<double> (
	unsigned blockSize,	unsigned gridSize, const hipStream_t &stream,
	const d_IntrlpGrid<double>& inner, const d_IntrlpGrid<double>& outer, const d_Protein<double>& prot,
	const unsigned& numDOFs,
	const double* data_in_x, const double* data_in_y, const double* data_in_z,
	double* data_out_x, double* data_out_y, double* data_out_z, double* data_out_E);

}  // namespace as


