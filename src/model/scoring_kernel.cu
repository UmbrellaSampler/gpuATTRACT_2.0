#include "hip/hip_runtime.h"
#include "scoring_kernel.h"

namespace as{

template<typename REAL, typename DOF_T >
__global__ void scoring_kernel(
		const d_IntrlpGrid<REAL> inner,
		const d_IntrlpGrid<REAL> outer,
		d_Protein<REAL> const  protein,
		DOF_T const * dofs,
		unsigned const numDOFs,
		unsigned const type_protein,
		REAL* buffer_defoX,
		REAL* buffer_defoY,
		REAL* buffer_defoZ,
		REAL* buffer_trafoX,
		REAL* buffer_trafoY,
		REAL* buffer_trafoZ,
		REAL* data_out_x,
		REAL* data_out_y,
		REAL* data_out_z,
		REAL* data_out_E)
{
	using real4_t = typename TypeWrapper<REAL>::real4_t;
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned numAtoms = protein.numAtoms;
	if (idx < numAtoms*numDOFs) {
		unsigned DOFidx = idx / numAtoms;
		auto dof = dofs[DOFidx];
		REAL x_trafo,y_trafo,z_trafo;
		float4 potForce;
		d_DOFPos_device( protein, dof, idx,	type_protein,
				 buffer_defoX[idx],  buffer_defoY[idx], buffer_defoZ[idx],
				 x_trafo,y_trafo,z_trafo
				);

		PotForce_device(inner, outer, protein, numDOFs, idx, x_trafo, y_trafo, z_trafo, potForce);
		 buffer_trafoX[idx] = x_trafo;
		 buffer_trafoY[idx] = y_trafo;
		 buffer_trafoZ[idx] = z_trafo;
	}
}


template<typename REAL, typename DOF_T>
 void d_score(
		 unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		const d_IntrlpGrid<REAL> inner,
		const d_IntrlpGrid<REAL> outer,
		d_Protein<REAL> const  protein,
		DOF_T const * dofs,
		unsigned const numDOFs,
		unsigned const type_protein,
		REAL* buffer_defoX,
		REAL* buffer_defoY,
		REAL* buffer_defoZ,
		REAL* buffer_trafoX,
		REAL* buffer_trafoY,
		REAL* buffer_trafoZ,
		REAL* data_out_x,
		REAL* data_out_y,
		REAL* data_out_z,
		REAL* data_out_E)
{
	cudaVerifyKernel((
			scoring_kernel<<<gridSize, blockSize, 0, stream>>> (
			inner,
			outer,
			protein,
			dofs,
			numDOFs,
			type_protein,
			buffer_defoX,
			buffer_defoY,
			buffer_defoZ,
			buffer_trafoX,
			buffer_trafoY,
			buffer_trafoZ,
			data_out_x,
			data_out_y,
			data_out_z,
			data_out_E))
		);
}
template
 void d_score<float, DOF_6D<float>>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		const d_IntrlpGrid<float> inner,
		const d_IntrlpGrid<float> outer,
		d_Protein<float> const  protein,
		DOF_6D<float> const * dofs,
		unsigned const numDOFs,
		unsigned const type_protein,
		float* buffer_defoX,
		float* buffer_defoY,
		float* buffer_defoZ,
		float* buffer_trafoX,
		float* buffer_trafoY,
		float* buffer_trafoZ,
		float* data_out_x,
		float* data_out_y,
		float* data_out_z,
		float* data_out_E);

template
 void d_score<double, DOF_6D<double>>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		const d_IntrlpGrid<double> inner,
		const d_IntrlpGrid<double> outer,
		d_Protein<double> const  protein,
		DOF_6D<double> const * dofs,
		unsigned const numDOFs,
		unsigned const type_protein,
		double* buffer_defoX,
		double* buffer_defoY,
		double* buffer_defoZ,
		double* buffer_trafoX,
		double* buffer_trafoY,
		double* buffer_trafoZ,
		double* data_out_x,
		double* data_out_y,
		double* data_out_z,
		double* data_out_E);

template
 void d_score<float, DOF_6D_Modes<float>>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		const d_IntrlpGrid<float> inner,
		const d_IntrlpGrid<float> outer,
		d_Protein<float> const  protein,
		DOF_6D_Modes<float> const * dofs,
		unsigned const numDOFs,
		unsigned const type_protein,
		float* buffer_defoX,
		float* buffer_defoY,
		float* buffer_defoZ,
		float* buffer_trafoX,
		float* buffer_trafoY,
		float* buffer_trafoZ,
		float* data_out_x,
		float* data_out_y,
		float* data_out_z,
		float* data_out_E);

template
 void d_score<double, DOF_6D_Modes<double>>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		const d_IntrlpGrid<double> inner,
		const d_IntrlpGrid<double> outer,
		d_Protein<double> const  protein,
		DOF_6D_Modes<double> const * dofs,
		unsigned const numDOFs,
		unsigned const type_protein,
		double* buffer_defoX,
		double* buffer_defoY,
		double* buffer_defoZ,
		double* buffer_trafoX,
		double* buffer_trafoY,
		double* buffer_trafoZ,
		double* data_out_x,
		double* data_out_y,
		double* data_out_z,
		double* data_out_E);

}
