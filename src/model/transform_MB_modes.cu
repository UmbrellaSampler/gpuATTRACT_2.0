#include "hip/hip_runtime.h"
#include "transform_MB_modes.h"

#include "Vec3.h"
#include "RotMat.h"
#include "matrixFunctions.h"
#include "macros.h"
namespace as {


/**
 * This function is ment for multi body transformation and deformation. It calculates all necessary positions for one ligand in relation to all other ligands in the system
 * and the receptor. Assuming that the partner of this ligand is located in the origin.
 *
 */
template<typename REAL>
__global__ void d_DOFPos(
		unsigned numLigands,
		unsigned ligIdx,
		REAL const* xLig,
		REAL const* yLig,
		REAL const* zLig,
		REAL const* xModesLig,
		REAL const* yModesLig,
		REAL const* zModesLig,
		DOF_6D_MB_Modes<REAL>* dofs,
		unsigned numAtomsLig,
		unsigned numModesLig,
		unsigned numDOFs,
		REAL* xLigDefo,
		REAL* yLigDefo,
		REAL* zLigDefo,
		REAL* xLigTrafo,
		REAL* yLigTrafo,
		REAL* zLigTrafo
		)
{
	/* calculate element index that is to be prcessed */
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;

	/* load DOF from global memory */
	unsigned DOFidx = idx / numAtomsLig;
	auto dof = dofs[DOFidx];
	unsigned atomIdx = idx % numAtomsLig;

		if (idx < numAtomsLig * numDOFs) {
			const RotMat<REAL> rotMat = euler2rotmat(dof._6D[ligIdx].ang.x, dof._6D[ligIdx].ang.y, dof._6D[ligIdx].ang.z);
			unsigned bufSize = numDOFs * numAtomsLig *numLigands;
			int bufIdx = numAtomsLig * DOFidx + atomIdx;

			Vec3<REAL> posAtomLig(xLig[atomIdx], yLig[atomIdx], zLig[atomIdx]);


			for(int mode=0; mode < numModesLig; mode++){
				posAtomLig.x += dof.modesLig[ligIdx][mode] * xModesLig[atomIdx*numModesLig+mode];
				posAtomLig.y += dof.modesLig[ligIdx][mode] * yModesLig[atomIdx*numModesLig+mode];
				posAtomLig.z += dof.modesLig[ligIdx][mode] * zModesLig[atomIdx*numModesLig+mode];
			}

			xLigDefo[bufIdx] = posAtomLig.x;
			yLigDefo[bufIdx] = posAtomLig.y;
			zLigDefo[bufIdx] = posAtomLig.z;


			for( int lig = 0; lig < numLigands; lig++){
				unsigned bufIdxMB = lig * bufSize + bufIdx;
				if( lig != ligIdx){
					//get the invers of the rotationmatrix of each ligand
					const RotMat<REAL> rotMatInv = euler2rotmat(dof._6D[lig].ang.x, dof._6D[lig].ang.y, dof._6D[lig].ang.z).getInv();
					//get the relative positon of ligand[ligIdx] to ligang[lig]
					Vec3<REAL> tRel =  dof._6D[ligIdx].pos - dof._6D[lig].pos;
					//rotate tRel into the system of ligand[lig]
					tRel = rotMatInv * tRel;
					//rotate each position of ligand[ligIdx] into the system of ligand[lig] and than rotate by the angle of ligand[ligIdx]
					posAtomLig = rotMatInv * posAtomLig;
					posAtomLig = rotMat *  posAtomLig;
					// add the relative translation with is now in the coordinate system of ligand[lig]
					posAtomLig += tRel;

					xLigTrafo[bufIdxMB] = posAtomLig.x;
					yLigTrafo[bufIdxMB] = posAtomLig.y;
					zLigTrafo[bufIdxMB] = posAtomLig.z;
				}
				//if lig == ligIdx calculate the orientation of ligand[ligIdx] relative to the receptor
				else if( lig == ligIdx){
					posAtomLig = rotMat*posAtomLig;
					posAtomLig += dof._6D[lig].pos;

					xLigTrafo[bufIdxMB] = posAtomLig.x;
					yLigTrafo[bufIdxMB] = posAtomLig.y;
					zLigTrafo[bufIdxMB] = posAtomLig.z;
				}
			}
		}

}


template<typename REAL>
__global__ void d_rotateForces(
		REAL* xForce,
		REAL* yForce,
		REAL* zForce,
		DOF_6D_MB_Modes<REAL>* dofs,
		unsigned numAtoms,
		unsigned numDOFs,
		unsigned ligIdx
)
{
	/* calculate element index that is to be prcessed */
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < numAtoms*numDOFs) {
		/* load DOF from global memory */
		unsigned DOFidx = idx / numAtoms;
		auto dof = dofs[DOFidx];
		unsigned atomIdx = idx % numAtoms;

		Vec3<REAL> ForceAtom(xForce[atomIdx], yForce[atomIdx], zForce[atomIdx]);
		const RotMat<REAL> rotMat = euler2rotmat(dof._6D[ligIdx].ang.x, dof._6D[ligIdx].ang.y, dof._6D[ligIdx].ang.z);

		ForceAtom=rotMat*ForceAtom;

		xForce[idx] = ForceAtom.x;
		yForce[idx] = ForceAtom.y;
		zForce[idx] = ForceAtom.z;
	}
}

template<typename REAL>
__global__ void d_DOFPos_receptor(
		unsigned numLigands,
		REAL const* xRec,
		REAL const* yRec,
		REAL const* zRec,
		REAL const* xModesRec,
		REAL const* yModesRec,
		REAL const* zModesRec,
		DOF_6D_MB_Modes<REAL>* dofs,
		unsigned numAtomsRec,
		unsigned numModesRec,
		unsigned numDOFs,
		REAL* xRecDefo,
		REAL* yRecDefo,
		REAL* zRecDefo,
		REAL* xRecTrafo,
		REAL* yRecTrafo,
		REAL* zRecTrafo
		)
{
	/* calculate element index that is to be prcessed */
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;



	if (idx < numAtomsRec * numDOFs) {
		unsigned DOFidx = idx / numAtomsRec;
		unsigned atomIdx = idx % numAtomsRec;
		unsigned bufSize = numAtomsRec * numLigands * numDOFs;
		int bufIdx = numAtomsRec * DOFidx + atomIdx;
		auto dof = dofs[DOFidx];


		Vec3<REAL> posAtomRec(xRec[bufIdx], yRec[bufIdx], zRec[bufIdx]);

		for(int mode=0; mode < numModesRec; mode++){
			posAtomRec.x += dof.modesRec[mode] * xModesRec[atomIdx*numModesRec+mode];
			posAtomRec.y += dof.modesRec[mode] * yModesRec[atomIdx*numModesRec+mode];
			posAtomRec.z += dof.modesRec[mode] * zModesRec[atomIdx*numModesRec+mode];
		}

		xRecDefo[bufIdx] = posAtomRec.x;
		yRecDefo[bufIdx] = posAtomRec.y;
		zRecDefo[bufIdx] = posAtomRec.z;
		for(unsigned ligIdx = 0; ligIdx < numLigands; ligIdx++){
			unsigned bufIdxMB = ligIdx * bufSize + bufIdx;
			const RotMat<REAL> rotMat = euler2rotmat(dof._6D[ligIdx].ang.x, dof._6D[ligIdx].ang.y, dof._6D[ligIdx].ang.z);
			const RotMat<REAL> rotMatInv = rotMat.getInv();
			Vec3<REAL> posInv = rotMatInv * dof._6D[ligIdx].pos.inv();

			posAtomRec = rotMatInv*posAtomRec;
			posAtomRec += posInv;

			xRecTrafo[bufIdxMB] = posAtomRec.x;
			yRecTrafo[bufIdxMB] = posAtomRec.y;
			zRecTrafo[bufIdxMB] = posAtomRec.z;
		}
	}

}


template<typename REAL>
void d_rotateForces(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		REAL* xForce,
		REAL* yForce,
		REAL* zForce,
		DOF_6D_MB_Modes<REAL>* dofs,
		unsigned numAtoms,
		unsigned numDOFs,
		unsigned ligIdx
)
{
	d_rotateForces<<<gridSize, blockSize, 0, stream>>> (
			xForce,
			yForce,
			zForce,
			dofs,
			numAtoms,
			numDOFs,
			ligIdx
			);
}




template<typename REAL>
void d_DOFPos(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		unsigned numLigands,
		unsigned ligIdx,
		REAL const* xLig,
		REAL const* yLig,
		REAL const* zLig,
		REAL const* xModesLig,
		REAL const* yModesLig,
		REAL const* zModesLig,
		DOF_6D_MB_Modes<REAL>* dofs,
		unsigned numAtomsLig,
		unsigned numModesLig,
		unsigned numDOFs,
		REAL* xLigDefo,
		REAL* yLigDefo,
		REAL* zLigDefo,
		REAL* xLigTrafo,
		REAL* yLigTrafo,
		REAL* zLigTrafo
		)
{
	cudaVerifyKernel((
			d_DOFPos<<<gridSize, blockSize, 0, stream>>> (
				numLigands,
				ligIdx,
				xLig,
				yLig,
				zLig,
				xModesLig,
				yModesLig,
				zModesLig,
				dofs,
				numAtomsLig,
				numModesLig,
				numDOFs,
				xLigDefo,
				yLigDefo,
				zLigDefo,
				xLigTrafo,
				yLigTrafo,
				zLigTrafo
				))
		);
}

template<typename REAL>
void d_DOFPos_receptor(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		unsigned numLigands,
		REAL const* xRec,
		REAL const* yRec,
		REAL const* zRec,
		REAL const* xModesRec,
		REAL const* yModesRec,
		REAL const* zModesRec,
		DOF_6D_MB_Modes<REAL>* dofs,
		unsigned numAtomsRec,
		unsigned numModesRec,
		unsigned numDOFs,
		REAL* xRecDefo,
		REAL* yRecDefo,
		REAL* zRecDefo,
		REAL* xRecTrafo,
		REAL* yRecTrafo,
		REAL* zRecTrafo
		)
{
	cudaVerifyKernel((
			d_DOFPos_receptor<<<gridSize, blockSize, 0, stream>>> (
				numLigands,
				xRec,
				yRec,
				zRec,
				xModesRec,
				yModesRec,
				zModesRec,
				dofs,
				numAtomsRec,
				numModesRec,
				numDOFs,
				xRecDefo,
				yRecDefo,
				zRecDefo,
				xRecTrafo,
				yRecTrafo,
				zRecTrafo
))
		);
}

template
void d_DOFPos<float>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		unsigned numLigands,
		unsigned ligIdx,
		float const* xLig,
		float const* yLig,
		float const* zLig,
		float const* xModesLig,
		float const* yModesLig,
		float const* zModesLig,
		DOF_6D_MB_Modes<float>* dofs,
		unsigned numAtomsLig,
		unsigned numModesLig,
		unsigned numDOFs,
		float* xLigDefo,
		float* yLigDefo,
		float* zLigDefo,
		float* xLigTrafo,
		float* yLigTrafo,
		float* zLigTrafo
		);

template
void d_DOFPos<double>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		unsigned numLigands,
		unsigned ligIdx,
		double const* xLig,
		double const* yLig,
		double const* zLig,
		double const* xModesLig,
		double const* yModesLig,
		double const* zModesLig,
		DOF_6D_MB_Modes<double>* dofs,
		unsigned numAtomsLig,
		unsigned numModesLig,
		unsigned numDOFs,
		double* xLigDefo,
		double* yLigDefo,
		double* zLigDefo,
		double* xLigTrafo,
		double* yLigTrafo,
		double* zLigTrafo
		);

template
void d_DOFPos_receptor<float>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		unsigned numLigands,
		float const* xRec,
		float const* yRec,
		float const* zRec,
		float const* xModesRec,
		float const* yModesRec,
		float const* zModesRec,
		DOF_6D_MB_Modes<float>* dofs,
		unsigned numAtomsRec,
		unsigned numModesRec,
		unsigned numDOFs,
		float* xRecDefo,
		float* yRecDefo,
		float* zRecDefo,
		float* xRecTrafo,
		float* yRecTrafo,
		float* zRecTrafo
		);

template
void d_DOFPos_receptor<double>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		unsigned numLigands,
		double const* xRec,
		double const* yRec,
		double const* zRec,
		double const* xModesRec,
		double const* yModesRec,
		double const* zModesRec,
		DOF_6D_MB_Modes<double>* dofs,
		unsigned numAtomsRec,
		unsigned numModesRec,
		unsigned numDOFs,
		double* xRecDefo,
		double* yRecDefo,
		double* zRecDefo,
		double* xRecTrafo,
		double* yRecTrafo,
		double* zRecTrafo
		);

template
void d_rotateForces<float>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		float* xForce,
		float* yForce,
		float* zForce,
		DOF_6D_MB_Modes<float>* dofs,
		unsigned numAtoms,
		unsigned numDOFs,
		unsigned ligIdx);

template
void d_rotateForces<double>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		double* xForce,
		double* yForce,
		double* zForce,
		DOF_6D_MB_Modes<double>* dofs,
		unsigned numAtoms,
		unsigned numDOFs,
		unsigned ligIdx);

}  // namespace as
