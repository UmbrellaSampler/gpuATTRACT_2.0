#include "hip/hip_runtime.h"
/*
 * neighborlist_MB_modes.cu
 *
 *  Created on: Sep 4, 2016
 *      Author: uwe
 */

#include "neighborlist_MB_modes.h"

#include "nativeTypesWrapper.h"
#include "DeviceNLGrid.h"
#include "DeviceProtein.h"
#include "DeviceParamTable.h"
#include "SimParam.h"
#include "forcefield.h"
#include "macros.h"


namespace as {
/*
 * In comparison to the NL force calculation without modes, this methode takes the deformed coordinates of the receptor instead of its original position, which does not contain mode deformation.
 * Furthermore the calculated force is rotated after calculation. This is important if we are calculating the forces acting on the receptor and are thus in the system of the ligand.
 * As final forces are always in the system of the receptor it is important to rotate the force back into the system of the receptor.
 * The receptor corresponds to the protein in which system we are in right now
 * RecPos corresponds to the deformed coordinates of the Protein.
 */
template<typename REAL>
__global__ void d_NLPotForce_rotate(
		const d_NLGrid<REAL> grid,
		const d_Protein<REAL> rec,
		const d_Protein<REAL> lig,
		const d_ParamTable<REAL> table,
		const SimParam<REAL> simParam,
		const DOF_6D_MB_Modes<REAL>* dofs,
		unsigned int ligIdx,
		const unsigned numDOFs,
		const REAL* RecPosX,
		const REAL* RecPosY,
		const REAL* RecPosZ,
		const REAL* LigPosX,
		const REAL* LigPosY,
		const REAL* LigPosZ,
		REAL* outLig_fx,
		REAL* outLig_fy,
		REAL* outLig_fz,
		REAL* outLigand_E)
{
	using real3_t = typename TypeWrapper<REAL>::real3_t;
	const unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned LigNumEl = lig.numAtoms;
	if (i < LigNumEl*numDOFs) {

		unsigned int DOFidx = i / LigNumEl;
		auto dof = dofs[DOFidx];
		const RotMat<REAL> rotMat = euler2rotmat(dof._6D[ligIdx].ang.x, dof._6D[ligIdx].ang.y, dof._6D[ligIdx].ang.z);

		const unsigned LigAttrIdx = i % LigNumEl;

		const unsigned atomTypeLig = lig.type[LigAttrIdx];

		if (atomTypeLig != 0) {


			const REAL posLigX = LigPosX[i];
			const REAL posLigY = LigPosY[i];
			const REAL posLigZ = LigPosZ[i];

			/* test if particle is out of bounds and perform data fetch and neigbourlist calculations */
			if (!(     (posLigX < grid.minDim.x || posLigX > grid.maxDim.x)
					|| (posLigY < grid.minDim.y || posLigY > grid.maxDim.y)
					|| (posLigZ < grid.minDim.z || posLigZ > grid.maxDim.z) ))
			{

				const uint2 nDesc = tex3D<uint2>(grid.tex,
						(posLigX - grid.minDim.x) * grid.dVox_inv + 0.5,
						(posLigY - grid.minDim.y) * grid.dVox_inv + 0.5,
						(posLigZ - grid.minDim.z) * grid.dVox_inv + 0.5);
				/* numEl = x; idx = y */


				real3_t fAcc = {0,0,0};
				REAL eAcc = 0;
				for (unsigned j = 0; j < nDesc.x; ++j) {
					const unsigned nIdx = grid.neighborList[nDesc.y + j];



					REAL dx = posLigX - RecPosX[nIdx];
					REAL dy = posLigY - RecPosX[nIdx];
					REAL dz = posLigZ - RecPosX[nIdx];
					const REAL dr2 = dx * dx + dy * dy + dz * dz;
					const REAL dPlateau2 = grid.dPlateau2;
					if ((dr2) > dPlateau2) {
						continue;
					}

					constexpr REAL one = static_cast<REAL>(1.0);
					const REAL dr2_inv = one/dr2; // inverse of dr2

					// Scale distances
					dx *= dr2_inv;
					dy *= dr2_inv;
					dz *= dr2_inv;

					real3_t fVdW;
					REAL eVdW;

					const size_t atomTypeRec = rec.type[nIdx];


					// calculate energy and potential/energy of LJ/VdW potential

					auto const params = table.getParams(atomTypeRec-1, atomTypeLig-1);
					LJPotForce(dr2, dr2_inv, dx, dy, dz,
							params,
							one, table.shape,
							fVdW.x, fVdW.y, fVdW.z, eVdW);

					fAcc.x  += fVdW.x;
					fAcc.y  += fVdW.y;
					fAcc.z  += fVdW.z;
					eAcc += eVdW;

					const REAL chargeLig = lig.charge[LigAttrIdx];
					const REAL chargeRec = rec.charge[nIdx];
					const REAL chargeLigRec = chargeLig * chargeRec * simParam.ffelec;

					const bool calc_elec = abs(chargeLigRec) > 0.001; // evaluate electric potential

					REAL dPlateau2_inv = 1/grid.dPlateau2;
					const REAL ratio = sqrt(dr2*dPlateau2_inv);
					REAL rdx = ratio*dx;
					REAL rdy = ratio*dy;
					REAL rdz = ratio*dz;

					LJPotForce(dPlateau2, dPlateau2_inv, rdx, rdy, rdz,
						params,
						one, table.shape,
						fVdW.x, fVdW.y, fVdW.z, eVdW);
					fAcc.x  -= fVdW.x;
					fAcc.y  -= fVdW.y;
					fAcc.z  -= fVdW.z;
					eAcc -= eVdW;


					if (calc_elec) {
						REAL eEl;
						real3_t fEl;

						// calculate energy and potential/energy of charge potential

						if (false) {
							printf("%u %f %f %f %u\n" ,
									i, posLigX, posLigY, posLigZ, atomTypeLig);
						}

						ChargePotForce(dr2_inv, dx, dy, dz,
								chargeLigRec,
								one, simParam.dielec,
								fEl.x, fEl.y, fEl.z, eEl);

						fAcc.x += fEl.x;
						fAcc.y += fEl.y;
						fAcc.z += fEl.z;
						eAcc += eEl;

						ChargePotForce(dPlateau2_inv, rdx, rdy, rdz,
								chargeLigRec,
								one, simParam.dielec,
								fEl.x, fEl.y, fEl.z, eEl);
						fAcc.x -= fEl.x;
						fAcc.y -= fEl.y;
						fAcc.z -= fEl.z;
						eAcc -= eEl;

					}
				}

				Vec3<REAL> force(fAcc.x, fAcc.y, fAcc.z);
				force = rotMat * force;
				/* store results back to global memory */
				if (nDesc.x > 0) {
					outLig_fx[i] += force.x;
					outLig_fy[i] += force.y;
					outLig_fz[i] += force.z;
					outLigand_E[i] += eAcc;
				}
			}
		} // if (atomtype != 0)
	}
}


template<typename REAL>
void d_NLPotForce(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		const d_NLGrid<REAL>& grid,
		const d_Protein<REAL>& rec,
		const d_Protein<REAL>& lig,
		const d_ParamTable<REAL>& table,
		const SimParam<REAL>& simParam,
		const DOF_6D_MB_Modes<REAL>* dofs,
		unsigned int ligIdx,
		const unsigned& numDOFs,
		const REAL* RecPosX,
		const REAL* RecPosY,
		const REAL* RecPosZ,
		const REAL* LigPosX,
		const REAL* LigPosY,
		const REAL* LigPosZ,
		REAL* outLig_fx,
		REAL* outLig_fy,
		REAL* outLig_fz,
		REAL* outLigand_E)
{
	cudaVerifyKernel((
			d_NLPotForce_rotate<<<gridSize, blockSize, 0, stream>>> (
				grid,
				rec,
				lig,
				table,
				simParam,
				dofs,
				ligIdx,
				numDOFs,
				RecPosX,
				RecPosY,
				RecPosZ,
				LigPosX,
				LigPosY,
				LigPosZ,
				outLig_fx,
				outLig_fy,
				outLig_fz,
				outLigand_E
			)
		));
}


template
void d_NLPotForce<float>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		const d_NLGrid<float>& grid,
		const d_Protein<float>& rec,
		const d_Protein<float>& lig,
		const d_ParamTable<float>& table,
		const SimParam<float>& simParam,
		const DOF_6D_MB_Modes<float>* dofs,
		unsigned int ligIdx,
		const unsigned& numDOFs,
		const float* RecPosX,
		const float* RecPosY,
		const float* RecPosZ,
		const float* LigPosX,
		const float* LigPosY,
		const float* LigPosZ,
		float* outLig_fx,
		float* outLig_fy,
		float* outLig_fz,
		float* outLigand_E
		);

template
void d_NLPotForce<double>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		const d_NLGrid<double>& grid,
		const d_Protein<double>& rec,
		const d_Protein<double>& lig,
		const d_ParamTable<double>& table,
		const SimParam<double>& simParam,
		const DOF_6D_MB_Modes<double>* dofs,
		unsigned int ligIdx,
		const unsigned& numDOFs,
		const double* RecPosX,
		const double* RecPosY,
		const double* RecPosZ,
		const double* LigPosX,
		const double* LigPosY,
		const double* LigPosZ,
		double* outLig_fx,
		double* outLig_fy,
		double* outLig_fz,
		double* outLigand_E
		);

}  // namespace as

