#include "hip/hip_runtime.h"
#include "transform_modes.h"

#include "Vec3.h"
#include "RotMat.h"
#include "matrixFunctions.h"
#include "macros.h"
namespace as {




template<typename REAL>
inline __device__ Vec3<REAL> invertDOF	(Vec3<REAL> posAtom,const RotMat<REAL> rotMat, Vec3<REAL> pos){
	const RotMat<REAL> rotMatInv = rotMat.getInv();
	Vec3<REAL> posInv = rotMatInv * pos.inv();
return posInv;
}


template<typename REAL>
inline __device__ void d_deform(Vec3<REAL>& posAtom,int atomIdx, REAL* dMode, REAL* xModes,REAL* yModes,REAL* zModes, int numModes){
	for(int mode=0; mode < numModes; mode++){
		posAtom.x += dMode[mode] * xModes[atomIdx*numModes+mode];
		posAtom.y += dMode[mode] * yModes[atomIdx*numModes+mode];
		posAtom.z += dMode[mode] * zModes[atomIdx*numModes+mode];
	}
}

template<typename REAL>
inline __device__ void d_translate_rotate	(Vec3<REAL>& posAtom, int atomIdx, Vec3<REAL> const& pos,const RotMat<REAL> rotMat){
	posAtom = rotMat*posAtom;
	posAtom += pos;
}


template<typename REAL>
__global__ void d_DOFPos(
		REAL const* xRec,
		REAL const* yRec,
		REAL const* zRec,
		REAL const* xLig,
		REAL const* yLig,
		REAL const* zLig,
		REAL const* xModesRec,
		REAL const* yModesRec,
		REAL const* zModesRec,
		REAL const* xModesLig,
		REAL const* yModesLig,
		REAL const* zModesLig,
		DOF_6D_Modes<REAL>* dofs,
		unsigned numAtomsRec,
		unsigned numAtomsLig,
		unsigned numModesRec,
		unsigned numModesLig,
		unsigned numDOFs,
		REAL* xRecDefo,
		REAL* yRecDefo,
		REAL* zRecDefo,
		REAL* xRecTrafo,
		REAL* yRecTrafo,
		REAL* zRecTrafo,
		REAL* xLigTrafo,
		REAL* yLigTrafo,
		REAL* zLigTrafo
		)
{
	/* calculate element index that is to be prcessed */
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int maxNumAtoms = max(numAtomsRec, numAtomsLig);

	if (idx < maxNumAtoms*numDOFs) {
		/* load DOF from global memory */
		unsigned DOFidx = idx / maxNumAtoms;
		auto dof = dofs[DOFidx];
		unsigned atomIdx = idx % maxNumAtoms;

		const RotMat<REAL> rotMat = euler2rotmat(dof._6D.ang.x, dof._6D.ang.y, dof._6D.ang.z);

		if (atomIdx < numAtomsRec){
			Vec3<REAL> posAtomRec(xRec[atomIdx], yRec[atomIdx], zRec[atomIdx]);

			for(int mode=0; mode < numModesRec; mode++){
				posAtomRec.x += dof.modesRec[mode] * xModesRec[atomIdx*numModesRec+mode];
				posAtomRec.y += dof.modesRec[mode] * yModesRec[atomIdx*numModesRec+mode];
				posAtomRec.z += dof.modesRec[mode] * zModesRec[atomIdx*numModesRec+mode];
			}

			xRecDefo[idx] = posAtomRec.x;
			yRecDefo[idx] = posAtomRec.y;
			zRecDefo[idx] = posAtomRec.z;

			const RotMat<REAL> rotMatInv = rotMat.getInv();
			Vec3<REAL> posInv = rotMatInv * dof._6D.pos.inv();
			posAtomRec = rotMat*posAtomRec;
			posAtomRec += dof._6D.pos;


			xRecTrafo[idx] = posAtomRec.x;
			yRecTrafo[idx] = posAtomRec.y;
			zRecTrafo[idx] = posAtomRec.z;
		}

		if (atomIdx < numAtomsLig){
			Vec3<REAL> posAtomLig(xLig[atomIdx], yLig[atomIdx], zLig[atomIdx]);


			for(int mode=0; mode < numModesLig; mode++){
				posAtomLig.x += dof.modesLig[mode] * xModesLig[atomIdx*numModesLig+mode];
				posAtomLig.y += dof.modesLig[mode] * yModesLig[atomIdx*numModesLig+mode];
				posAtomLig.z += dof.modesLig[mode] * zModesLig[atomIdx*numModesLig+mode];
			}
			posAtomLig = rotMat*posAtomLig;
			posAtomLig += dof._6D.pos;
			xLigTrafo[idx] = posAtomLig.x;
			yLigTrafo[idx] = posAtomLig.y;
			zLigTrafo[idx] = posAtomLig.z;
		}
	}
}


template<typename REAL>
__global__ void d_rotateForces(
		REAL* xForce,
		REAL* yForce,
		REAL* zForce,
		DOF_6D_Modes<REAL>* dofs,
		unsigned numAtoms,
		unsigned numDOFs
)
{
	/* calculate element index that is to be prcessed */
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < numAtoms*numDOFs) {
		/* load DOF from global memory */
		unsigned DOFidx = idx / numAtoms;
		auto dof = dofs[DOFidx];
		unsigned atomIdx = idx % numAtoms;

		Vec3<REAL> ForceAtom(xForce[atomIdx], yForce[atomIdx], zForce[atomIdx]);
		const RotMat<REAL> rotMat = euler2rotmat(dof._6D.ang.x, dof._6D.ang.y, dof._6D.ang.z);

		ForceAtom=rotMat*ForceAtom;

		xForce[idx] = ForceAtom.x;
		yForce[idx] = ForceAtom.y;
		zForce[idx] = ForceAtom.z;
	}
}



template<typename REAL>
void d_rotateForces(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		REAL* xForce,
		REAL* yForce,
		REAL* zForce,
		DOF_6D_Modes<REAL>* dofs,
		unsigned numAtoms,
		unsigned numDOFs
)
{
	d_rotateForces<<<gridSize, blockSize, 0, stream>>> (
			xForce,
			yForce,
			zForce,
			dofs,
			numAtoms,
			numDOFs
			);
}




template<typename REAL>
void d_DOFPos(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		REAL const* xRec,
		REAL const* yRec,
		REAL const* zRec,
		REAL const* xLig,
		REAL const* yLig,
		REAL const* zLig,
		REAL const* xModesRec,
		REAL const* yModesRec,
		REAL const* zModesRec,
		REAL const* xModesLig,
		REAL const* yModesLig,
		REAL const* zModesLig,
		DOF_6D_Modes<REAL>* dofs,
		unsigned numAtomsRec,
		unsigned numAtomsLig,
		unsigned numModesRec,
		unsigned numModesLig,
		unsigned numDOFs,
		REAL* xRecDefo,
		REAL* yRecDefo,
		REAL* zRecDefo,
		REAL* xRecTrafo,
		REAL* yRecTrafo,
		REAL* zRecTrafo,
		REAL* xLigTrafo,
		REAL* yLigTrafo,
		REAL* zLigTrafo)
{
	cudaVerifyKernel((
			d_DOFPos<<<gridSize, blockSize, 0, stream>>> (
			xRec,
			yRec,
			zRec,
			xLig,
			yLig,
			zLig,
			xModesRec,
			yModesRec,
			zModesRec,
			xModesLig,
			yModesLig,
			zModesLig,
			dofs,
			numAtomsRec,
			numAtomsLig,
			numModesRec,
			numModesLig,
			numDOFs,
			xRecDefo,
			yRecDefo,
			zRecDefo,
			xRecTrafo,
			yRecTrafo,
			zRecTrafo,
			xLigTrafo,
			yLigTrafo,
			zLigTrafo
			))
		);
}

template
void d_DOFPos<float>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		float const* xRec,
		float const* yRec,
		float const* zRec,
		float const* xLig,
		float const* yLig,
		float const* zLig,
		float const* xModesRec,
		float const* yModesRec,
		float const* zModesRec,
		float const* xModesLig,
		float const* yModesLig,
		float const* zModesLig,
		DOF_6D_Modes<float>* dofs,
		unsigned numAtomsRec,
		unsigned numAtomsLig,
		unsigned numModesRec,
		unsigned numModesLig,
		unsigned numDOFs,
		float* xRecDefo,
		float* yRecDefo,
		float* zRecDefo,
		float* xRecTrafo,
		float* yRecTrafo,
		float* zRecTrafo,
		float* xLigTrafo,
		float* yLigTrafo,
		float* zLigTrafo);

template
void d_DOFPos<double>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		double const* xRec,
		double const* yRec,
		double const* zRec,
		double const* xLig,
		double const* yLig,
		double const* zLig,
		double const* xModesRec,
		double const* yModesRec,
		double const* zModesRec,
		double const* xModesLig,
		double const* yModesLig,
		double const* zModesLig,
		DOF_6D_Modes<double>* dofs,
		unsigned numAtomsRec,
		unsigned numAtomsLig,
		unsigned numModesRec,
		unsigned numModesLig,
		unsigned numDOFs,
		double* xRecDefo,
		double* yRecDefo,
		double* zRecDefo,
		double* xRecTrafo,
		double* yRecTrafo,
		double* zRecTrafo,
		double* xLigTrafo,
		double* yLigTrafo,
		double* zLigTrafo);

}  // namespace as
