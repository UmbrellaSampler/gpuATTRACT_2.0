#include "hip/hip_runtime.h"
#include "transform_modes.h"

#include "Vec3.h"
#include "RotMat.h"
#include "matrixFunctions.h"
#include "macros.h"

namespace as {


template <typename REAL, typename DOF_T>
__device__ void deform( DOF_T const& dof, Vec3<REAL> & posAtom, d_Protein<REAL> const*  protein, unsigned const idxAtom, unsigned const idxModes, unsigned const bufIdx,
		REAL* buffer_defoX, REAL* buffer_defoY, REAL* buffer_defoZ,
		typename std::enable_if<std::is_same< DOF_T, DOF_6D_Modes<REAL> >::value, void>::type* dummy = 0 )
{
	unsigned const numModes = protein->numModes;
	for(int mode=0; mode < numModes; mode++){
		posAtom.x += dof.modesRec[mode] * protein->xModes[idxAtom*numModes+mode];
		posAtom.y += dof.modesRec[mode] * protein->yModes[idxAtom*numModes+mode];
		posAtom.z += dof.modesRec[mode] * protein->zModes[idxAtom*numModes+mode];
	}
		buffer_defoX[bufIdx] = posAtom.x;
		buffer_defoZ[bufIdx] = posAtom.y;
		buffer_defoZ[bufIdx] = posAtom.z;

}

template <typename REAL, typename DOF_T>
__device__ void deform( DOF_T const& dof, Vec3<REAL> & posAtom, d_Protein<REAL> const*  protein, unsigned const idxAtom, unsigned const idxModes, unsigned const bufIdx,
		REAL* buffer_defoX, REAL* buffer_defoY, REAL* buffer_defoZ,
		typename std::enable_if<std::is_same< DOF_T, DOF_6D<REAL> >::value, void>::type* dummy = 0 )
{

}


template <typename REAL, typename DOF_T, int PROTEIN_T>
__device__ void translate_rotate( DOF_T const& dof, Vec3<REAL> & posAtom,
	typename std::enable_if<std::is_same< DOF_T, DOF_6D_Modes<REAL> >::value, void>::type* dummy = 0 )
{
	RotMat<REAL> rotMat = euler2rotmat(dof._6D.ang.x, dof._6D.ang.y, dof._6D.ang.z);
	if ( PROTEIN_T == 0){
		rotMat = rotMat.getInv();
	}

	posAtom = rotMat*posAtom;
	posAtom += dof._6D.pos;
}

template <typename REAL, typename DOF_T, int PROTEIN_T>
__device__ void translate_rotate( DOF_T const& dof, Vec3<REAL> & posAtom ,
	typename std::enable_if<std::is_same< DOF_T, DOF_6D<REAL> >::value, void>::type* dummy = 0 )
{
	RotMat<REAL> rotMat = euler2rotmat(dof.ang.x, dof.ang.y, dof.ang.z);
	if ( PROTEIN_T == 0){
		rotMat = rotMat.getInv();
	}
	posAtom = rotMat*posAtom;
	posAtom += dof.pos;
}

template<typename REAL, typename DOF_T, int PROTEIN_T >
__global__ void d_DOFPos_kernel(
		d_Protein<REAL> const*  protein,
		DOF_T* dofs,
		unsigned const numDOFs,
		 REAL* buffer_defoX, REAL* buffer_defoY, REAL* buffer_defoZ,
		 REAL* buffer_trafoX, REAL* buffer_trafoY, REAL* buffer_trafoZ
		)
{
	/* calculate element index that is to be prcessed */
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int num_atoms = protein->numAtoms;

	if (idx < num_atoms*numDOFs) {
		/* load DOF from global memory */
		unsigned DOFidx = idx / num_atoms;
		auto dof = dofs[DOFidx];
		unsigned atomIdx = idx % num_atoms;

		Vec3<REAL> posAtom(	protein->xPos[atomIdx],
							protein->xPos[atomIdx],
							protein->xPos[atomIdx]);

		deform< REAL, DOF_T>( dof, posAtom, protein, atomIdx, 0, idx, buffer_defoX, buffer_defoY, buffer_defoZ);

		translate_rotate< REAL, DOF_T, PROTEIN_T>( dof, posAtom );

		buffer_trafoX[idx] = posAtom.x;
		buffer_trafoY[idx] = posAtom.y;
		buffer_trafoZ[idx] = posAtom.z;

	}
}



template<typename REAL>
__global__ void d_rotateForces(
		REAL* xForce, REAL* yForce, REAL* zForce,
		DOF_6D_Modes<REAL>* dofs,
		unsigned const numAtoms,
		unsigned const numDofs
)
{
	/* calculate element index that is to be prcessed */
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;

		if (idx < numAtoms*numDofs) {
		/* load DOF from global memory */
		unsigned DOFidx = idx / numAtoms;
		auto dof = dofs[DOFidx];


		Vec3<REAL> ForceAtom(xForce[idx], yForce[idx], zForce[idx]);
		const RotMat<REAL> rotMat = euler2rotmat(dof._6D.ang.x, dof._6D.ang.y, dof._6D.ang.z);

		ForceAtom = rotMat*ForceAtom;

		xForce[idx] = ForceAtom.x;
		yForce[idx] = ForceAtom.y;
		zForce[idx] = ForceAtom.z;
	}
}



template<typename REAL>
void d_rotateForces(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		REAL* xForce,
		REAL* yForce,
		REAL* zForce,
		DOF_6D_Modes<REAL>* dofs,
		unsigned numAtoms,
		unsigned numDOFs
)
{
	d_rotateForces<<<gridSize, blockSize, 0, stream>>> (
			xForce,
			yForce,
			zForce,
			dofs,
			numAtoms,
			numDOFs
			);
}




template<typename REAL, typename DOF_T, int PROTEIN_T >
 void d_DOFPos(
		 unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		d_Protein<REAL> const*  protein,
		DOF_T* dofs,
		unsigned const numDOFs,
		REAL* buffer_defoX, REAL* buffer_defoY, REAL* buffer_defoZ,
		 REAL* buffer_trafoX, REAL* buffer_trafoY, REAL* buffer_trafoZ
		)
{
	cudaVerifyKernel((
		d_DOFPos_kernel< REAL, DOF_T,  PROTEIN_T ><<<gridSize, blockSize, 0, stream>>> (
		blockSize,
		gridSize,
		stream,
		protein,
		dofs,
		numDOFs,
		buffer_defoX,  buffer_defoY,  buffer_defoZ,
		buffer_trafoX,  buffer_trafoY,  buffer_trafoZ
		))
		);
}


template
__global__ void d_DOFPos_kernel<float, DOF_6D_Modes<float>, 0>(

		d_Protein<float> const*  protein,
		DOF_6D_Modes<float>* dofs,
		unsigned const numDOFs,
		float* buffer_defoX, float* buffer_defoY, float* buffer_defoZ,
		float* buffer_trafoX, float* buffer_trafoY, float* buffer_trafoZ
		);

template
__global__ void d_DOFPos_kernel<double, DOF_6D_Modes<double>, 0>(

		d_Protein<double> const*  protein,
		DOF_6D_Modes<double>* dofs,
		unsigned const numDOFs,
		double* buffer_defoX, double* buffer_defoY, double* buffer_defoZ,
		double* buffer_trafoX, double* buffer_trafoY, double* buffer_trafoZ
		);

template
__global__ void d_DOFPos_kernel<float, DOF_6D_Modes<float>, 1>(

		d_Protein<float> const*  protein,
		DOF_6D_Modes<float>* dofs,
		unsigned const numDOFs,
		float* buffer_defoX, float* buffer_defoY, float* buffer_defoZ,
		float* buffer_trafoX, float* buffer_trafoY, float* buffer_trafoZ
		);

template
__global__ void d_DOFPos_kernel<double, DOF_6D_Modes<double>, 1>(

		d_Protein<double> const*  protein,
		DOF_6D_Modes<double>* dofs,
		unsigned const numDOFs,
		double* buffer_defoX, double* buffer_defoY, double* buffer_defoZ,
		double* buffer_trafoX, double* buffer_trafoY, double* buffer_trafoZ
		);

template
__global__ void d_DOFPos_kernel<float, DOF_6D<float>, 0>(

		d_Protein<float> const*  protein,
		DOF_6D<float>* dofs,
		unsigned const numDOFs,
		float* buffer_defoX, float* buffer_defoY, float* buffer_defoZ,
		float* buffer_trafoX, float* buffer_trafoY, float* buffer_trafoZ
		);

template
__global__ void d_DOFPos_kernel<double, DOF_6D<double>, 0>(

		d_Protein<double> const*  protein,
		DOF_6D<double>* dofs,
		unsigned const numDOFs,
		double* buffer_defoX, double* buffer_defoY, double* buffer_defoZ,
		double* buffer_trafoX, double* buffer_trafoY, double* buffer_trafoZ
		);

template
__global__ void d_DOFPos_kernel<float, DOF_6D<float>, 1>(

		d_Protein<float> const*  protein,
		DOF_6D<float>* dofs,
		unsigned const numDOFs,
		float* buffer_defoX, float* buffer_defoY, float* buffer_defoZ,
		float* buffer_trafoX, float* buffer_trafoY, float* buffer_trafoZ
		);

template
__global__ void d_DOFPos_kernel<double, DOF_6D<double>, 1>(

		d_Protein<double> const*  protein,
		DOF_6D<double>* dofs,
		unsigned const numDOFs,
		double* buffer_defoX, double* buffer_defoY, double* buffer_defoZ,
		double* buffer_trafoX, double* buffer_trafoY, double* buffer_trafoZ
		);

/*
template
 void d_DOFPos<float, DOF_6D_Modes<float>, 0>(
		 unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		d_Protein<float> const*  protein,
		DOF_6D_Modes<float>* dofs,
		unsigned const numDOFs,
		float* buffer_defoX, float* buffer_defoY, float* buffer_defoZ,
		float* buffer_trafoX, float* buffer_trafoY, float* buffer_trafoZ
		);

template
 void d_DOFPos<double, DOF_6D_Modes<double>, 0>(
		 unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		d_Protein<double> const*  protein,
		DOF_6D_Modes<double>* dofs,
		unsigned const numDOFs,
		double* buffer_defoX, double* buffer_defoY, double* buffer_defoZ,
		double* buffer_trafoX, double* buffer_trafoY, double* buffer_trafoZ
		);

template
 void d_DOFPos<float, DOF_6D_Modes<float>, 1>(
		 unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		d_Protein<float> const*  protein,
		DOF_6D_Modes<float>* dofs,
		unsigned const numDOFs,
		float* buffer_defoX, float* buffer_defoY, float* buffer_defoZ,
		float* buffer_trafoX, float* buffer_trafoY, float* buffer_trafoZ
		);

template
 void d_DOFPos<double, DOF_6D_Modes<double>, 1>(
		 unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		d_Protein<double> const*  protein,
		DOF_6D_Modes<double>* dofs,
		unsigned const numDOFs,
		double* buffer_defoX, double* buffer_defoY, double* buffer_defoZ,
		double* buffer_trafoX, double* buffer_trafoY, double* buffer_trafoZ
		);

template
 void d_DOFPos<float, DOF_6D<float>, 0>(
		 unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		d_Protein<float> const*  protein,
		DOF_6D<float>* dofs,
		unsigned const numDOFs,
		float* buffer_defoX, float* buffer_defoY, float* buffer_defoZ,
		float* buffer_trafoX, float* buffer_trafoY, float* buffer_trafoZ
		);

template
 void d_DOFPos<double, DOF_6D<double>, 0>(
		 unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		d_Protein<double> const*  protein,
		DOF_6D<double>* dofs,
		unsigned const numDOFs,
		double* buffer_defoX, double* buffer_defoY, double* buffer_defoZ,
		double* buffer_trafoX, double* buffer_trafoY, double* buffer_trafoZ
		);

template
 void d_DOFPos<float, DOF_6D<float>, 1>(
		 unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		d_Protein<float> const*  protein,
		DOF_6D<float>* dofs,
		unsigned const numDOFs,
		float* buffer_defoX, float* buffer_defoY, float* buffer_defoZ,
		float* buffer_trafoX, float* buffer_trafoY, float* buffer_trafoZ
		);

template
 void d_DOFPos<double, DOF_6D<double>, 1>(
		 unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		d_Protein<double> const*  protein,
		DOF_6D<double>* dofs,
		unsigned const numDOFs,
		double* buffer_defoX, double* buffer_defoY, double* buffer_defoZ,
		double* buffer_trafoX, double* buffer_trafoY, double* buffer_trafoZ
		);


*/




template
void d_rotateForces<float>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		float* xForce,
		float* yForce,
		float* zForce,
		DOF_6D_Modes<float>* dofs,
		unsigned numAtoms,
		unsigned numDOFs);

template
void d_rotateForces<double>(
		unsigned blockSize,
		unsigned gridSize,
		const hipStream_t &stream,
		double* xForce,
		double* yForce,
		double* zForce,
		DOF_6D_Modes<double>* dofs,
		unsigned numAtoms,
		unsigned numDOFs);

}  // namespace as
